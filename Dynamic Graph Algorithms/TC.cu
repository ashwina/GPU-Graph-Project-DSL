#include "hip/hip_runtime.h"
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include "../graph.hpp"
#include "../libcuda.cuh"

// kernel for computing TC
// edges are u->v, v->w & w->u
__global__ void tc_kernel(int V, int *d_meta, int *d_data, unsigned long long *d_Tcount)
{
  unsigned int u = blockDim.x * blockIdx.x + threadIdx.x;
  if (u < V)
  {
    for (int idx1 = d_meta[u]; idx1 < d_meta[u + 1]; idx1++)
    {
      int v = d_data[idx1];
      if (u < v)
      {
        for (int idx2 = d_meta[v]; idx2 < d_meta[v + 1]; idx2++)
        {
          int w = d_data[idx2];
          if (v < w)
          {
            // check for w->u edge
            for (int idx3 = d_meta[w]; idx3 < d_meta[w + 1]; idx3++)
            {
              if (d_data[idx3] == u)
                atomicAdd(d_Tcount, 1);
            }
          }
        }
      }
    }
  }
}

// kernel for computing the number of triangles added/removed as edges get added/deleted
__global__ void count_triangles(int batchSize, int E, update *d_batch, bool *d_modified,
                                int *d_meta, int *d_data, int *d_diffMeta, int *d_diffData,
                                unsigned long long *d_count1, unsigned long long *d_count2, unsigned long long *d_count3)
{
  unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < batchSize)
  {
    int u = d_batch[id].source;
    int v = d_batch[id].destination;

    for (int idx1 = d_meta[v]; idx1 < d_meta[v + 1]; idx1++)
    {
      int w = d_data[idx1];
      if (w != INT_MAX && u != v && u != w && v != w)
      {
        int count = 1;
        bool isTriangle = false;
        if (d_modified[idx1])
          count += 1;

        if (__isEdge(w, u, d_meta, d_data, d_diffMeta, d_diffData))
        {
          isTriangle = true;
          int idx2 = __getEdge(w, u, E, d_meta, d_data, d_diffMeta, d_diffData);
          if (d_modified[idx2])
            count += 1;
        }

        if (isTriangle)
        {
          if (count == 1)
            atomicAdd(d_count1, 1);
          else if (count == 2)
            atomicAdd(d_count2, 1);
          else if (count == 3)
            atomicAdd(d_count3, 1);
        }
      }
    }

    for (int idx1 = d_diffMeta[v]; idx1 < d_diffMeta[v + 1]; idx1++)
    {
      int w = d_diffData[idx1];
      if (w != INT_MAX && u != v && u != w && v != w)
      {
        int count = 1;
        bool isTriangle = false;
        if (d_modified[E + idx1])
          count += 1;

        if (__isEdge(w, u, d_meta, d_data, d_diffMeta, d_diffData))
        {
          isTriangle = true;
          int idx2 = __getEdge(w, u, E, d_meta, d_data, d_diffMeta, d_diffData);
          if (d_modified[idx2])
            count += 1;
        }

        if (isTriangle)
        {
          if (count == 1)
            atomicAdd(d_count1, 1);
          else if (count == 2)
            atomicAdd(d_count2, 1);
          else if (count == 3)
            atomicAdd(d_count3, 1);
        }
      }
    }
  }
}

// check for hipError_t
void checkCudaError()
{
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
  {
    printf("CUDA error: %s\n", hipGetErrorString(error));
  }
}

unsigned long long Compute_TC(graph &g)
{
  // data on host
  int V = g.num_nodes();
  int E = g.num_edges();

  int *h_meta = g.indexofNodes;
  int *h_data = g.edgeList;
  unsigned long long h_Tcount = 0;

  // data on device
  int *d_meta;
  int *d_data;
  unsigned long long *d_Tcount;

  // for recording the total time taken
  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // allocate memory on device
  hipMalloc(&d_meta, sizeof(int) * (V + 1));
  hipMalloc(&d_data, sizeof(int) * (E));
  hipMalloc(&d_Tcount, sizeof(unsigned long long));

  // copy to device
  hipMemcpy(d_meta, h_meta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_data, h_data, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_Tcount, &h_Tcount, sizeof(unsigned long long), hipMemcpyHostToDevice);

  // launch config
  const int numThreads = 1024;
  const int numBlocks = (V + numThreads - 1) / numThreads;

  // call kernel to compute the triangle counts
  tc_kernel<<<numBlocks, numThreads>>>(V, d_meta, d_data, d_Tcount);
  hipDeviceSynchronize();
  checkCudaError();

  // copy distances back to host
  hipMemcpy(&h_Tcount, d_Tcount, sizeof(unsigned long long), hipMemcpyDeviceToHost);

  // print time taken
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Initial graph, TC using GPU - time taken: %.3f ms \n", milliseconds);

  // free up the memory
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_Tcount);

  return h_Tcount;
}

unsigned long long Compute_dynamicTCDel(graph &g, unsigned long long triangleCount,
                                        bool *modified, std::vector<update> &deleteBatch)
{
  // data on host
  int V = g.num_nodes();
  int E = g.num_edges_CSR();
  int diffE = g.num_edges_diffCSR();
  int batchSize = deleteBatch.size();

  int *h_meta = g.indexofNodes;
  int *h_data = g.edgeList;
  int *h_diffMeta = g.diff_indexofNodes;
  int *h_diffData = g.diff_edgeList;

  unsigned long long h_Tcount = triangleCount;
  unsigned long long h_count1, h_count2, h_count3;
  h_count1 = h_count2 = h_count3 = 0;

  // data on device
  update *d_deleteBatch;
  bool *d_modified;
  int *d_meta;
  int *d_data;
  int *d_diffMeta;
  int *d_diffData;
  unsigned long long *d_count1, *d_count2, *d_count3;

  // for recording the total time taken
  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // allocate memory on device
  hipMalloc(&d_deleteBatch, sizeof(update) * (batchSize));
  hipMalloc(&d_modified, sizeof(bool) * (E + diffE));
  hipMalloc(&d_meta, sizeof(int) * (V + 1));
  hipMalloc(&d_data, sizeof(int) * (E));
  hipMalloc(&d_diffMeta, sizeof(int) * (V + 1));
  hipMalloc(&d_diffData, sizeof(int) * (diffE));
  hipMalloc(&d_count1, sizeof(unsigned long long));
  hipMalloc(&d_count2, sizeof(unsigned long long));
  hipMalloc(&d_count3, sizeof(unsigned long long));

  // copy to device
  hipMemcpy(d_deleteBatch, deleteBatch.data(), sizeof(update) * (batchSize), hipMemcpyHostToDevice);
  hipMemcpy(d_modified, modified, sizeof(bool) * (E + diffE), hipMemcpyHostToDevice);
  hipMemcpy(d_meta, h_meta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_data, h_data, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_diffMeta, h_diffMeta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_diffData, h_diffData, sizeof(int) * (diffE), hipMemcpyHostToDevice);
  hipMemcpy(d_count1, &h_count1, sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMemcpy(d_count2, &h_count2, sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMemcpy(d_count3, &h_count3, sizeof(unsigned long long), hipMemcpyHostToDevice);

  // launch config
  const int numThreads = 1024;
  const int numBlocks = (batchSize + numThreads - 1) / numThreads;

  // call kernel to compute the reduction in number of triangles
  count_triangles<<<numBlocks, numThreads>>>(batchSize, E, d_deleteBatch, d_modified,
                                             d_meta, d_data, d_diffMeta, d_diffData,
                                             d_count1, d_count2, d_count3);
  hipDeviceSynchronize();
  checkCudaError();

  // copy to host
  hipMemcpy(&h_count1, d_count1, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy(&h_count2, d_count2, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy(&h_count3, d_count3, sizeof(unsigned long long), hipMemcpyDeviceToHost);

  // print the time taken
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Compute DynTC-Del - time taken: %.3f ms \n", milliseconds);

  // free up the memory
  hipFree(d_deleteBatch);
  hipFree(d_modified);
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_diffMeta);
  hipFree(d_diffData);
  hipFree(d_count1);
  hipFree(d_count2);
  hipFree(d_count3);

  h_Tcount = h_Tcount - (h_count1 / 2 + h_count2 / 4 + h_count3 / 6);
  return h_Tcount;
}

unsigned long long Compute_dynamicTCAdd(graph &g, unsigned long long triangleCount,
                                        bool *modified, std::vector<update> &addBatch)
{
  // data on host
  int V = g.num_nodes();
  int E = g.num_edges_CSR();
  int diffE = g.num_edges_diffCSR();
  int batchSize = addBatch.size();

  int *h_meta = g.indexofNodes;
  int *h_data = g.edgeList;
  int *h_diffMeta = g.diff_indexofNodes;
  int *h_diffData = g.diff_edgeList;

  unsigned long long h_Tcount = triangleCount;
  unsigned long long h_count1, h_count2, h_count3;
  h_count1 = h_count2 = h_count3 = 0;

  // data on device
  update *d_addBatch;
  bool *d_modified;
  int *d_meta;
  int *d_data;
  int *d_diffMeta;
  int *d_diffData;
  unsigned long long *d_count1, *d_count2, *d_count3;

  // for recording the total time taken
  float milliseconds = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // allocate memory on device
  hipMalloc(&d_addBatch, sizeof(update) * (batchSize));
  hipMalloc(&d_modified, sizeof(bool) * (E + diffE));
  hipMalloc(&d_meta, sizeof(int) * (V + 1));
  hipMalloc(&d_data, sizeof(int) * (E));
  hipMalloc(&d_diffMeta, sizeof(int) * (V + 1));
  hipMalloc(&d_diffData, sizeof(int) * (diffE));
  hipMalloc(&d_count1, sizeof(unsigned long long));
  hipMalloc(&d_count2, sizeof(unsigned long long));
  hipMalloc(&d_count3, sizeof(unsigned long long));

  // copy to device
  hipMemcpy(d_addBatch, addBatch.data(), sizeof(update) * (batchSize), hipMemcpyHostToDevice);
  hipMemcpy(d_modified, modified, sizeof(bool) * (E + diffE), hipMemcpyHostToDevice);
  hipMemcpy(d_meta, h_meta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_data, h_data, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(d_diffMeta, h_diffMeta, sizeof(int) * (V + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_diffData, h_diffData, sizeof(int) * (diffE), hipMemcpyHostToDevice);
  hipMemcpy(d_count1, &h_count1, sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMemcpy(d_count2, &h_count2, sizeof(unsigned long long), hipMemcpyHostToDevice);
  hipMemcpy(d_count3, &h_count3, sizeof(unsigned long long), hipMemcpyHostToDevice);

  // launch config
  const int numThreads = 1024;
  const int numBlocks = (batchSize + numThreads - 1) / numThreads;

  // call kernel to compute the number of triangles new triangles
  count_triangles<<<numBlocks, numThreads>>>(batchSize, E, d_addBatch, d_modified,
                                             d_meta, d_data, d_diffMeta, d_diffData,
                                             d_count1, d_count2, d_count3);
  hipDeviceSynchronize();
  checkCudaError();

  // copy to host
  hipMemcpy(&h_count1, d_count1, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy(&h_count2, d_count2, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  hipMemcpy(&h_count3, d_count3, sizeof(unsigned long long), hipMemcpyDeviceToHost);

  // print the time taken
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Compute DynTC-Add - time taken: %.3f ms \n", milliseconds);

  // free up the memory
  hipFree(d_addBatch);
  hipFree(d_modified);
  hipFree(d_meta);
  hipFree(d_data);
  hipFree(d_diffMeta);
  hipFree(d_diffData);
  hipFree(d_count1);
  hipFree(d_count2);
  hipFree(d_count3);

  h_Tcount = h_Tcount + (h_count1 / 2 + h_count2 / 4 + h_count3 / 6);
  return h_Tcount;
}

int main(int argc, char **argv)
{
  graph G("../inputGraphs/inputs/usaRoadNet.txt", "cuda", false);
  G.parseGraph();

  unsigned long long triangleCount = Compute_TC(G);
  printf("Initial TC = %llu\n", triangleCount);

  bool *modified_add;
  bool *modified_del;
  int elements = 0;

  std::vector<update> updateEdges = G.parseUpdates("../inputGraphs/updates/update_usaRoadNet.txt");
  std::vector<update> deleteBatch;
  std::vector<update> addBatch;

  int percent = atoi(argv[1]);
  int updateSize = percent * 0.01 * G.num_edges();
  if (!G.isGraphDirected() && updateSize & 1)
    updateSize++;
  int batchSize = updateSize;
  // int batchSize = 1000000;

  std::chrono::time_point<std::chrono::high_resolution_clock> startTime, endTime;
  std::chrono::duration<double, std::milli> timeTaken;

  startTime = std::chrono::high_resolution_clock::now();
  for (int k = 0; k < updateSize; k += batchSize)
  {
    if ((k + batchSize) > updateSize)
      elements = updateSize - k;
    else
      elements = batchSize;

    // batch-wise deletion and decremental call
    modified_del = new bool[G.num_edges()];
    for (int i = 0; i < G.num_edges(); i++)
      modified_del[i] = false;

    for (int i = k; i < (k + batchSize) && i < updateSize; i++)
    {
      if (updateEdges[i].type == 'd')
      {
        update u = updateEdges[i];
        int src = u.source;
        int dest = u.destination;
        for (edge e : G.getNeighbors(src))
        {
          if (e.destination == dest)
            modified_del[e.id] = true;
        }
        deleteBatch.push_back(u);
      }
    }

    triangleCount = Compute_dynamicTCDel(G, triangleCount, modified_del, deleteBatch);
    G.updateCSR_Del(updateEdges, k, elements);
    deleteBatch.clear();

    // batch-wise addition and incremental call
    G.updateCSR_Add(updateEdges, k, elements);

    modified_add = new bool[G.num_edges()];
    for (int i = 0; i < G.num_edges(); i++)
      modified_add[i] = false;

    for (int i = k; i < (k + batchSize) && i < updateSize; i++)
    {
      if (updateEdges[i].type == 'a')
      {
        update u = updateEdges[i];
        int src = u.source;
        int dest = u.destination;
        for (edge e : G.getNeighbors(src))
        {
          if (e.destination == dest)
          {
            modified_add[e.id] = true;
          }
        }
        addBatch.push_back(u);
      }
    }

    triangleCount = Compute_dynamicTCAdd(G, triangleCount, modified_add, addBatch);
    addBatch.clear();

    free(modified_add);
    free(modified_del);
  }
  endTime = std::chrono::high_resolution_clock::now();
  timeTaken = endTime - startTime;
  printf("Total time taken for %d percent updates: %.3f ms\n", percent, timeTaken.count());
  printf("Final TC = %llu\n\n", triangleCount);

  return 0;
}

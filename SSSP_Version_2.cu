#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "graph.hpp"

#include <sys/time.h>



template <typename T>
__global__ void initKernel(unsigned nSize,T* dArray,T initVal){
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id < nSize){
    dArray[id]=initVal;
  }
}


__global__ void SSSP_Kernel(int * gpu_OA , int * gpu_edgeList , int* weight, int * gpu_dist , int src ,int V, int MAX_VAL , bool * gpu_modified_prev,
bool * gpu_modified_next, bool * gpu_finished) {

unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
 
if (id < V) {

if(gpu_modified_prev[id]) {
 for (int edge = gpu_OA[id]; edge < gpu_OA[id+1]; edge ++) 
 {
          int nbr = gpu_edgeList[edge] ;
          int e = edge;
          int dist_new;
          
          dist_new = gpu_dist[id] + weight[e];
           
          if ( gpu_dist[id] != MAX_VAL  && gpu_dist[nbr] > dist_new)
          {
               atomicMin(&gpu_dist[nbr] , dist_new);
               gpu_modified_next[nbr]=true;
               gpu_finished[0] = false;
            
          }
   }
   }
   }

} 

void SSSP(int * OA , int * edgeList , int* cpu_edgeLen , int * dist , int src ,int V, int E )
{
  int MAX_VAL = 2147483647 ;
  
  struct timeval t1, t2;
  
  clock_t start , end;
  double gpu_time_use;
  
  int *gpu_edgeList;
  int *gpu_edgeLen;
  int *gpu_dist;
  int *gpu_OA;
  bool *gpu_modified_prev;
  bool *gpu_modified_next;
  bool * gpu_finished;

  hipMalloc( &gpu_OA, sizeof(int) * (1+V) );
  hipMalloc( &gpu_edgeList, sizeof(int) * (E) );
  hipMalloc( &gpu_edgeLen, sizeof(int) * (E) );
  hipMalloc( &gpu_dist, sizeof(int) * (V) );
  hipMalloc( &gpu_modified_prev, sizeof(bool) * (V) );
  hipMalloc( &gpu_modified_next, sizeof(bool) * (V) );
  hipMalloc( &gpu_finished, sizeof(bool) * (1) );
  
  unsigned int block_size;
	unsigned int num_blocks;
 
 
  
  if(V <= 1024)
	{
		block_size = V;
		num_blocks = 1;
	}
	else
	{
		block_size = 1024;
		num_blocks = ceil(((float)V) / block_size);
			
	}
  
  bool* modified= new bool[V];
  bool* finished = new bool[1]; 
  
  

  for (int t = 0; t < V; t ++) 
 {
    dist[t] = INT_MAX;
    modified[t] = false;
  }
  
  
  modified[src] = true;
  dist[src] = 0;
  finished[0] = false;
  
  hipMemcpy(gpu_OA, OA, sizeof(int) * (1+V), hipMemcpyHostToDevice);
  hipMemcpy(gpu_edgeList, edgeList, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(gpu_edgeLen,cpu_edgeLen , sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(gpu_dist,dist , sizeof(int) * (V), hipMemcpyHostToDevice);
  hipMemcpy(gpu_modified_prev,modified , sizeof(bool) * (V), hipMemcpyHostToDevice);
  hipMemcpy(gpu_finished,finished , sizeof(bool) * (1), hipMemcpyHostToDevice);
  
  int k =0;
  while ( !finished[0] )
  {
     
      SSSP_Kernel<<<num_blocks , block_size>>>(gpu_OA,gpu_edgeList, gpu_edgeLen ,gpu_dist,src, V ,MAX_VAL , gpu_modified_prev, gpu_modified_next, gpu_finished);
        
      
      
    initKernel<bool><<<num_blocks, block_size>>>(V, gpu_modified_prev, false);
    hipDeviceSynchronize();
  
    bool *tempModPtr  = gpu_modified_next;
    gpu_modified_next = gpu_modified_prev;
    gpu_modified_prev = tempModPtr;
    ++k;
    
    if(k==V){
      break;
    }
    
    }
  
   hipMemcpy(dist,gpu_dist , sizeof(int) * (V), hipMemcpyDeviceToHost);
   
   
   printf("\n");
   
   
   for (int i = 0; i <V; i++)
   {
      printf("%d  %d\n", i, dist[i]);
   }
   
 
  char *outputfilename = "output.txt";
  FILE *outputfilepointer;
  outputfilepointer = fopen(outputfilename, "w");
  
  fprintf(outputfilepointer, "%d\n", gpu_time_use);


  for (int i = 0; i <V; i++)
  {
    fprintf(outputfilepointer, "%d  %d\n", i, dist[i]);
  }
 
  
}

// driver program to test above function
int main(int argc , char ** argv)
{
  graph G("/home/ashwina/soc-LiveJournal1.txt");
  G.parseGraph();
   
  int V = G.num_nodes();
//---------------------------------------//   
   //printf("number of nodes =%d\n",V);
//-------------------------------------// 
 int E = G.num_edges();
 
 //---------------------------------------//   
  // printf("number of edges =%d\n",E);
//-------------------------------------// 
  
  int* edgeLen = G.getEdgeLen();
  
  int* dist;

  int src=0;
  
  int *OA;
  int *edgeList;
  int *cpu_edgeLen;
  
  
   OA = (int *)malloc( (V+1)*sizeof(int));
   edgeList = (int *)malloc( (E)*sizeof(int));
   cpu_edgeLen = (int *)malloc( (E)*sizeof(int));
   dist = (int *)malloc( (V)*sizeof(int));
  
  for(int i=0; i<= V; i++) {
    int temp = G.indexofNodes[i];
    OA[i] = temp;
  }
  
  for(int i=0; i< E; i++) {
    int temp = G.edgeList[i];
    edgeList[i] = temp;
  }
  
  for(int i=0; i< E; i++) {
    int temp = edgeLen[i];
    cpu_edgeLen[i] = temp;
  }
  
  
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    SSSP(OA,edgeList, cpu_edgeLen ,dist,src, V,E);
    hipDeviceSynchronize();
  
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
 

  return 0;

}

#include "hip/hip_runtime.h"
#include <limits.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "graph.hpp"


__global__ void SSSP_Kernel(int * gpu_OA , int * gpu_edgeList , int* weight, int * gpu_dist , int src ,int V, int MAX_VAL) {

unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
 
 if (id ==0 || id ==1) {
 for (int edge = gpu_OA[id]; edge < gpu_OA[id+1]; edge ++) 
 {
          int nbr = gpu_edgeList[edge] ;
          int e = edge;
          int dist_new;
          
           dist_new = gpu_dist[id] + weight[e];
          
          if ( gpu_dist[id] != MAX_VAL  && gpu_dist[nbr] > dist_new)
          {
            //bool modified_new = true;
           // omp_set_lock(&(lock[nbr])) ;
           // if (gpu_dist[nbr] > dist_new)
            //{
            
               atomicMin(&gpu_dist[nbr] , dist_new);
               
              // printf("%d  %d\n", gpu_dist[nbr],dist_new);
             
             
              //gpu_dist[nbr] = dist_new;
              //modified[nbr] = modified_new;
            //}
            //omp_unset_lock(&(lock[nbr]));
            
          }
   }

   }

} 

void SSSP(int * OA , int * edgeList , int* cpu_edgeLen , int * dist , int src ,int V, int E)
{
  int MAX_VAL = 2147483647 ;
  
  int *gpu_edgeList;
  int *gpu_edgeLen;
  int *gpu_dist;
  int *gpu_OA;

  hipMalloc( &gpu_OA, sizeof(int) * (1+V) );
  hipMalloc( &gpu_edgeList, sizeof(int) * (E) );
  hipMalloc( &gpu_edgeLen, sizeof(int) * (E) );
  hipMalloc( &gpu_dist, sizeof(int) * (V) );
  
  unsigned int block_size;
	unsigned int num_blocks;
 
 
  
  if(V <= 1024)
	{
		block_size = V;
		num_blocks = 1;
	}
	else
	{
		block_size = 1024;
		num_blocks = ceil(((float)V) / block_size);
			
	}
  
  bool* modified=new bool[V];
  
  

  for (int t = 0; t < V; t ++) 
 {
    dist[t] = INT_MAX;
    modified[t] = false;
  }
  
  
  modified[src] = true;
  dist[src] = 0;
  bool finished = false;
  
  hipMemcpy(gpu_OA, OA, sizeof(int) * (1+V), hipMemcpyHostToDevice);
  hipMemcpy(gpu_edgeList, edgeList, sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(gpu_edgeLen,cpu_edgeLen , sizeof(int) * (E), hipMemcpyHostToDevice);
  hipMemcpy(gpu_dist,dist , sizeof(int) * (V), hipMemcpyHostToDevice);
  
  
 // while ( !finished )
 // {
      //if (modified[id] == true ){
      //  modified[index] = false;
      
      for (int i = 1; i <= V - 1; i++) 
        {
            SSSP_Kernel<<<num_blocks , block_size>>>(gpu_OA,gpu_edgeList, gpu_edgeLen ,gpu_dist,src, V ,MAX_VAL);
            hipDeviceSynchronize();
        }
      
   // }
    //bool modified_fp = false ;
   // for (int v = 0; v < V; v ++) 
    //modified_fp = modified_fp || modified[v] ;
    //finished = !modified_fp ;
  //}
  
  
  
   hipMemcpy(dist,gpu_dist , sizeof(int) * (V), hipMemcpyDeviceToHost);
   
   printf("\n");
   
   
   for (int i = 0; i <V; i++)
   {
      printf("%d  %d\n", i, dist[i]);
   }
 
  char *outputfilename = "outputSG.txt";
  FILE *outputfilepointer;
  outputfilepointer = fopen(outputfilename, "w");


  for (int i = 0; i <V; i++)
  {
    fprintf(outputfilepointer, "%d  %d\n", i, dist[i]);
  }
 
  
}

// driver program to test above function
int main(int argc , char ** argv)
{
  graph G("/home/ashwina/cuda/final/input.txt");
  G.parseGraph();
   
  int V = G.num_nodes();
//---------------------------------------//   
   //printf("number of nodes =%d\n",V);
//-------------------------------------// 
 int E = G.num_edges();
 
 //---------------------------------------//   
  // printf("number of edges =%d\n",E);
//-------------------------------------// 
  
  int* edgeLen = G.getEdgeLen();
  
  int* dist;

  int src=0;
  
  int *OA;
  int *edgeList;
  int *cpu_edgeLen;
  
  
   OA = (int *)malloc( (V+1)*sizeof(int));
   edgeList = (int *)malloc( (E)*sizeof(int));
   cpu_edgeLen = (int *)malloc( (E)*sizeof(int));
   dist = (int *)malloc( (V)*sizeof(int));
  
  for(int i=0; i<= V; i++) {
    int temp = G.indexofNodes[i];
    OA[i] = temp;
  }
  
  for(int i=0; i< E; i++) {
    int temp = G.edgeList[i];
    edgeList[i] = temp;
  }
  
  for(int i=0; i< E; i++) {
    int temp = edgeLen[i];
    cpu_edgeLen[i] = temp;
  }

  SSSP(OA,edgeList, cpu_edgeLen ,dist,src, V,E);
 

  return 0;

}
